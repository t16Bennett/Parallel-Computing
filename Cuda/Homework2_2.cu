/*
	Homework Assignment 2-2
	Thomas Bennett
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

using namespace std;

__global__ void transpose(char* matrix, char* tMatrix, int m, int n){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<(m*n)){
		tMatrix[i] = matrix[(i/m)+(i%m)*n];
	}
}

int main(int argc, char** argv){
	char * file = argv[1];
	FILE * myfile;
	long lSize;
	char * buffer;
	size_t result;
	myfile = fopen(file, "rb");
	if(myfile!=NULL){
		fseek(myfile, 0, SEEK_END);
		lSize = ftell(myfile);
		rewind(myfile);
		buffer = (char *) malloc (sizeof(char)*lSize);
		result = fread(buffer,1,lSize,myfile);
		fclose(myfile);
		
		int n = (int) buffer[0] - 48;
		int m = (int) buffer[2] - 48;
		char* matrix;
		matrix = (char *) malloc (sizeof(char)*m*n);
		for(int i = 0; i<m*n;i++){
			matrix[i] = buffer[i*2+4];
		}

		char* tMatrix = (char*)malloc(sizeof(char)*m*n);
		char* gmatrix;
		hipMalloc(&gmatrix,sizeof(char)*m*n);
		char* gtMatrix;
		hipMalloc(&gtMatrix,sizeof(char)*m*n);
		hipMemcpy(gmatrix,matrix,sizeof(char)*m*n, hipMemcpyHostToDevice);
		int dimBlock = n;
		int dimGrid = m;
		transpose<<<dimGrid, dimBlock>>>(gmatrix,gtMatrix,m,n);

		hipMemcpy(tMatrix, gtMatrix, sizeof(char)*m*n, hipMemcpyDeviceToHost);
		for(int i = 0; i < n; i++){
			for(int j = 0; j < m; j++){
				cout << tMatrix[i*m+j] << " ";
			}
			cout << endl;
		}
		
		hipFree(gmatrix);
		hipFree(gtMatrix);
		free(buffer);
		free(matrix);
		free(tMatrix);
	}
	else cout << "Unable to open file";
	return 0;
}
