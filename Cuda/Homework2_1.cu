/*
	Homework 2 Part 1
	Thomas Bennett
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

using namespace std;

__global__ void findOnes(char* matrix, float* count, int N){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < N){
		if(matrix[i]=='1'){
			atomicAdd(&count[0], 1.0f);	
		}
	}	
}

int main(int argc, char **argv){
	char * file = argv[1];
	FILE * myfile;
	long lSize;
	char * buffer;
	size_t result; 
	myfile = fopen(file, "rb");
	if(myfile!=NULL){
		fseek(myfile, 0, SEEK_END);
		lSize = ftell(myfile);
		rewind(myfile);
		buffer = (char *) malloc (sizeof(char)*lSize);
		result = fread(buffer,1,lSize,myfile);
		fclose(myfile);
		int n = (int) buffer[0] - 48;
		int m = (int) buffer[2] - 48; 
		char* matrix;
		matrix = (char *) malloc (sizeof(char)*m*n);
		for(int i = 0; i<m*n;i++){
			matrix[i] = buffer[i*2+4];
			//cout << matrix[i] << endl;
		}
		float* count = (float*)malloc(sizeof(float));
		char* gmatrix;
		hipMalloc(&gmatrix, sizeof(char)*m*n);
		float* gcount;
		hipMalloc(&gcount, sizeof(float));
		hipMemcpy(gmatrix, matrix, sizeof(char)*m*n, hipMemcpyHostToDevice);
		int dimBlock = n;
		int dimGrid = m;
		findOnes<<<dimGrid, dimBlock>>>(gmatrix,gcount,m*n);
		hipMemcpy(count, gcount, sizeof(float), hipMemcpyDeviceToHost);
		cout << count[0] << endl;
		hipFree(gmatrix);
		hipFree(gcount);
		free(buffer);
		free(count);
		free(matrix);
	}
	else cout << "Unable to open file";
	return 0;
}
